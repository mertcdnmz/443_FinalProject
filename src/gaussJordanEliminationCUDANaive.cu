#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include "util.h"
using namespace std;


// This kernel function takes the current pivot value and divide all indices of row with this value to make diagonels(pivots) 1 
__global__ void MakePivotsOne(double* matrix, int rowSize, int colSize, int currCol, double pivotValue)
{
    
	int tID = currCol * colSize + threadIdx.x ;
	if ( tID < rowSize * colSize  )
	{
        matrix[tID] = matrix[tID] / pivotValue;
	}
    __syncthreads();
}

// This kernel function makes all top and bottom values of pivot's column zero
__global__ void MakePivotsColumnZero(double* matrix, int rowSize, int colSize, int currCol) 
{
    
    int index = threadIdx.x ;
    int currRow = blockIdx.x;
    int tID = currRow * colSize +  index;
 
    double rateWithPivot = matrix[ currRow * colSize + currCol ] ;
    __syncthreads();

    int colNumber = tID % colSize;
     
    if(currRow != currCol && index < colSize ){
        matrix[tID] = matrix[tID] - (rateWithPivot * matrix[( currCol * colSize ) + colNumber ]);
        __syncthreads();
    }
}

__global__ void PrintMatrixGPU(double *a, int n)
{   
    printf("\n") ;
    for (int i = 0; i < n -1   ; i++) 
    {
        for (int j = 0; j < n; j++){
            printf("%.3f ",  a[ i * n + j ]) ;
        }
        
        printf("\n")  ;
    }
    printf("\n") ;
}

int main(int argc, char const *argv[]) {

{
    //printing device properties
    int nDevices;
    hipGetDeviceCount(&nDevices);
    printf("***************Device Properties****************\n");
    for (int i = 0; i < nDevices; i++) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("Device name: %s\n", prop.name);
        printf("Max Grid Size X: %d\n", prop.maxGridSize[0]);
        printf("Max Grid Size Y: %d\n", prop.maxGridSize[1]);
        printf("Max Grid Size Z: %d\n", prop.maxGridSize[2]);
        printf("Max Number of Threads X: %d\n", prop.maxThreadsDim[0]);
        printf("Max Number of Threads Y: %d\n", prop.maxThreadsDim[1]);
        printf("Max Number of Threads Z: %d\n", prop.maxThreadsDim[2]);
        printf("Warp size: %d\n\n", prop.warpSize);
    }

}   

    int rowSize = stoi(argv[1]);
    int colSize = rowSize + 1 ;
    size_t size = rowSize * colSize * sizeof(double);
  
    double *h_Matrix  = (double*) malloc(size);
    createMatrix(h_Matrix, rowSize, true);
    
    hipError_t err = hipSuccess; // error handling
    
    // Allocate the device input matrix 
    double *d_Matrix = NULL;
    err = hipMalloc(&d_Matrix, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy matrix values to GPU memory
    err = hipMemcpy(d_Matrix, h_Matrix, size, hipMemcpyHostToDevice);

    printf("Gauss Jordan Elimination method calculation is started! \n");

    clock_t start_time = clock();
    for(int currCol = 0; currCol < rowSize  ; currCol++ )
    {
        dim3 gridShape(1,1) ;
        dim3 blockShape(colSize,1 ) ; //multi dimensional 
        double currentPivotValue = h_Matrix[ currCol * colSize + currCol ];
        MakePivotsOne<<<gridShape, blockShape>>>(d_Matrix, rowSize, colSize, currCol, currentPivotValue);
        hipDeviceSynchronize();

		// Check for errors  
		err = hipGetLastError();
		if (err != hipSuccess) 
		{
			std::cout << "Kernel failed: " << hipGetErrorString(err) << std::endl;
			hipFree(d_Matrix);

			return false;
		}
        
        MakePivotsColumnZero<<<rowSize, colSize>>>(d_Matrix, rowSize, colSize, currCol);
        hipDeviceSynchronize();
        err = hipMemcpy(h_Matrix, d_Matrix,size, hipMemcpyDeviceToHost); // copy device matrix into host matrix to take correct pivot value 
        
		// Check for errors
		err = hipGetLastError();
		if (err != hipSuccess) 
		{
			std::cout << "Kernel failed: " << hipGetErrorString(err) << std::endl;
			hipFree(d_Matrix);
			return false;
		}

    }
    
    clock_t total_time = clock() - start_time; 


    // Free device global memory
    err = hipFree(d_Matrix);
    if (err != hipSuccess)
    {   
        fprintf(stderr, "Failed to free device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Gauss Jordan Elimination method calculation is finished! \n\n");
    float total_ms =  float( total_time) * 1000 / CLOCKS_PER_SEC;
    printf("Time taken %f milliseconds for %d x %d matrix with naive CUDA implementation!\n\n",  total_ms, rowSize, rowSize);

    saveMatrix(h_Matrix,rowSize);
    checkMatrix(rowSize);
    
    // Free host memory
    free(h_Matrix);

    printf("******************Program Finished!*****************\n");
return 0;
}